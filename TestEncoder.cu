
#include <hip/hip_runtime.h>
#include <random>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <errno.h>

void ldpcEncoder (unsigned int *infoWord, unsigned int* W_ROW_ROM,
                  unsigned int numMsgBits, unsigned int numRowsinRom, unsigned int numParBits,
                  unsigned int shiftRegLength,
                  unsigned int *codeWord);

int main (int argc, char **argv) {

  unsigned int numChecks, numBits, maxBitsForCheck, maxChecksForBit;
  unsigned int  *mapRows2Cols, *mapCols2Rows;
  unsigned int numRowsW, numColsW, numParityBits, shiftRegLength;
  unsigned int *W_ROW_ROM;

  char mapFile[256];
  char wROM_File[256];
  FILE *src;
  int errnum;
  unsigned int infoLeng, rnum, rdenom;

  unsigned int  seed = 163331;
  /*  or use this to get a fresh sequence each time the program is run.
  std::random_device  rd;  //Will be used to obtain a seed for the random number engine
  std::mt19937 generator(rd()); //Standard mersenne_twister_engine seeded with rd()
  */
  std::mt19937 generator(seed); //Standard mersenne_twister_engine
  std::uniform_real_distribution<> rDist(0, 1);

  if (argc < 4) {
    printf("usage:  TestEncoder <infoLength> <r-numerator> <r-denominator>\n" );
    exit(-1);
  }
  infoLeng = atoi(argv[1]);
  rnum = atoi(argv[2]);
  rdenom = atoi(argv[3]);
  sprintf(mapFile, "./G_and_H_Matrices/Maps_%d%d_%d.bin", rnum, rdenom, infoLeng);
  sprintf(wROM_File, "./G_and_H_Matrices/W_ROW_ROM_%d%d_%d.bin", rnum, rdenom, infoLeng);

  src = fopen(mapFile, "r");
  if (src == NULL) {
    errnum = errno;
    printf("Value of errno: %d\n", errnum);
    perror("Error printed by perror");
    printf("Error opening file %s\n", mapFile);
    return(EXIT_FAILURE);
  }

  fread(& numBits, sizeof(unsigned int), 1, src);
  fread(& numChecks, sizeof(unsigned int), 1, src);
  fread(& maxBitsForCheck, sizeof(unsigned int), 1, src);
  fread(& maxChecksForBit, sizeof(unsigned int), 1, src);

  // These maps have an extra column (+1),
  // since each row begins with the actual length for the row.
  mapCols2Rows = (unsigned int*) malloc(numBits * (maxChecksForBit +1) * sizeof( unsigned int));
  mapRows2Cols = (unsigned int*) malloc(numChecks * (maxBitsForCheck +1) * sizeof( unsigned int));

  fread(mapCols2Rows, sizeof(unsigned int), numBits* (maxChecksForBit+1), src);
  fread(mapRows2Cols, sizeof(unsigned int), numChecks* (maxBitsForCheck+1), src);
  fclose(src);

  src = fopen(wROM_File, "r");
  if (src == NULL) {
    errnum = errno;
    printf("Value of errno: %d\n", errnum);
    perror("Error printed by perror");
    printf("Error opening file %s\n", mapFile);
    return(EXIT_FAILURE);
  }

  fread(& numRowsW, sizeof(unsigned int), 1, src);
  fread(& numColsW, sizeof(unsigned int), 1, src);
  fread(& shiftRegLength, sizeof(unsigned int), 1, src);
  W_ROW_ROM = (unsigned int*) malloc(numRowsW * numColsW * sizeof( unsigned int));
  fread(W_ROW_ROM, sizeof(unsigned int), numRowsW * numColsW, src);
  numParityBits = numColsW;
  fclose(src);

  printf("parameters have been read.\n");
  printf("numBits = %i, numChecks = %i\n", numBits, numChecks);
  printf("Max checks for bit: %i  Max bits for check %i\n", maxChecksForBit, maxBitsForCheck);
  // ///////////////////////////////////////////

  unsigned int* infoWord;
  unsigned int* codeWord;

  infoWord = (unsigned int *)malloc(infoLeng * sizeof(unsigned int));
  codeWord = (unsigned int *)malloc(numBits * sizeof(unsigned int));

    for (unsigned int j=0; j < infoLeng; j++) {
      infoWord[j] = (0.5 >= rDist(generator))? 1:0;
    }
    ldpcEncoder(infoWord, W_ROW_ROM, infoLeng, numRowsW, numColsW, shiftRegLength, codeWord);

    for (unsigned int j=0; j< numParityBits; j++) {
      printf(" %i", codeWord[infoLeng+j]);
      if ( (j % 40) == 39)  { printf("\n"); }
    }
    printf("\n");
}
