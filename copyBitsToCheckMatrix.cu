#include "hip/hip_runtime.h"
#include "bundleElt.h"

// copyBitsToCheckmatrix accepts a vector of the current bitEstimates
// and copies them into a checkRow matrix, where each row represents a check.
// It also generates a HardDecision copy of that output matrix checkRows.
__global__ void
copyBitsToCheckmatrix (unsigned int* map, bundleElt *bitEstimates, bundleElt *checkRows,
                       unsigned int numBits, unsigned int maxChecksForBit) {
  // index
  unsigned int m, n;
  unsigned int thisRowLength;
  unsigned int cellIndex, oneDindex;
  bundleElt thisBitEstimate;

  n = blockIdx.x;
  m = threadIdx.x + 1;
  if (n < numBits) {
    thisRowLength = map[n];
    thisBitEstimate = bitEstimates[n];
    if (m <= thisRowLength) {
      cellIndex = m * numBits + n;
      oneDindex = map[cellIndex];
      checkRows[oneDindex] = thisBitEstimate;
    }
  }
}
