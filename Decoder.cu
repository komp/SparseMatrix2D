#include "hip/hip_runtime.h"
// #define INTERNAL_TIMINGS_4_DECODER ON

// Based on the Eric's C-code implementation of ldpc decode.
//
#include <math.h>
#include <string.h>
#include <time.h>

#include "GPUincludes.h"

// #define MIN(a,b)  (((a) < (b)) ? (a) : (b))
#define ABS(a)  (((a) < (0)) ? (-(a)) : (a))
#define MAX_ETA                1e6
#define SCALE_FACTOR           0.75

#define NTHREADS    128

__global__ void
checkNodeProcessing (unsigned int numChecks, unsigned int maxBitsForCheck,
                      // eta is IN and OUT
                      float *lambdaByCheckIndex, float *eta) {
  // edk  HACK !!!
  // This was signs[maxBitsForCheck], which generates the error:
  // error: constant value is not known.
  // Since we are in a kernel function, we probably need a compile-time constant.
  // 128 should be much larger than maxBitsForCheck for any reasonable LDPC encoding.
  unsigned int signs[128];
  unsigned int signProduct;
  float value, min1, min2;
  unsigned int minIndex;

  // index
  unsigned int m;
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int thisRowLength, thisRowStart, currentIndex;

  if (tid < numChecks) {
    m = tid;
    thisRowStart = m * (maxBitsForCheck+1);
    // signs[n]  == 0  ==>  positive; 1  ==>  negative
    memset(signs, 0, (maxBitsForCheck+1)*sizeof(signs[0]));
    signProduct = 0;
    min1 = MAX_ETA;
    min2 =  MAX_ETA;
    minIndex = 1;
    thisRowLength = eta[thisRowStart];
    for (unsigned int n=1; n<= thisRowLength ; n++) {
      currentIndex = thisRowStart+n;
      value = eta[currentIndex] - lambdaByCheckIndex[currentIndex];
      signs[n] = (value < 0)? 1 : 0;
      signProduct = (signProduct != signs[n])? 1 : 0;
      value = ABS(value);
      if (value < min1) {
        min2 = min1;
        min1 = value;
        minIndex = n;
      } else if ( value < min2) {
        min2 = value;
      }
    }
    min1 = min1 * SCALE_FACTOR * (-1);
    min2 = min2 * SCALE_FACTOR * (-1);
    for (unsigned int n=1; n<= thisRowLength; n++) {
      currentIndex = thisRowStart+n;
      eta[currentIndex] =  (n == minIndex) ? min2 : min1;
      if (signs[n] != signProduct) {eta[currentIndex] = -eta[currentIndex];}
    }
  }
}

__global__ void
bitEstimates(float *rSig, float *etaByBitIndex, float *lambda,
             unsigned int numBits, unsigned int maxChecksForBit) {

  unsigned int n;
  unsigned int thisRowLength, thisRowStart;
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < numBits) {
    n = tid;
    float sum = rSig[n];
    thisRowStart = n*(maxChecksForBit+1);
    thisRowLength = etaByBitIndex[thisRowStart];
    for (unsigned int m=1; m<=thisRowLength; m++) {
      sum = sum + etaByBitIndex[thisRowStart +m];
    }
    lambda[n] = sum;
  }
}

int ldpcDecoder (float *rSig, unsigned int numChecks, unsigned int numBits,
                 unsigned int maxBitsForCheck, unsigned int maxChecksForBit,
                 unsigned int *mapRows2Cols,
                 unsigned int *mapCols2Rows,
                 unsigned int maxIterations,
                 unsigned int *decision,
                 float *estimates) {

  unsigned int nChecksByBits = numChecks*(maxBitsForCheck+1);
  unsigned int nBitsByChecks = numBits*(maxChecksForBit+1);

  float eta[nChecksByBits];
  float lambda[numBits];
  float etaByBitIndex[nBitsByChecks];
  float lambdaByCheckIndex[nChecksByBits];
  unsigned int cHat [nChecksByBits];

  unsigned int iterCounter;
  bool allChecksPassed = false;

  unsigned int oneDindex;
  unsigned int rowStart;
  unsigned int rowLength;

  float *dev_rSig;
  float *dev_eta;
  float *dev_lambda;
  float *dev_etaByBitIndex;
  float *dev_lambdaByCheckIndex;

  HANDLE_ERROR( hipMalloc( (void**)&dev_rSig, numBits * sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&dev_eta, nChecksByBits * sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&dev_lambda, numBits * sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&dev_etaByBitIndex,  nBitsByChecks * sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&dev_lambdaByCheckIndex, nChecksByBits * sizeof(float) ) );
  //  HANDLE_ERROR( hipMalloc( (void**)&dev_cHat, nChecksByBits * sizeof(unsigned int) ) );

  memcpy(lambda, rSig, numBits*sizeof(lambda[0]));
  memset(eta, 0, nChecksByBits*sizeof(eta[0]));
  memset(lambdaByCheckIndex, 0, nChecksByBits*sizeof(eta[0]));

  // Need to insert rowLengths into eta (and lambdaByCheckIndex)
  // with rows corresponding to parity checks.
  rowStart = 0;
  for (unsigned int check=0; check<numChecks; check++) {
    rowLength = mapRows2Cols[rowStart];
    eta[rowStart] = (float)rowLength;
    lambdaByCheckIndex[rowStart] = (float)rowLength;
    cHat[rowStart] = rowLength;
    rowStart = rowStart + (maxBitsForCheck+1);
  }

  // Need to insert rowLengths into etaByBitIndex
  rowStart = 0;
  for (unsigned int bit=0; bit<numBits; bit++) {
    etaByBitIndex[rowStart] = (float)mapCols2Rows[rowStart];
    rowStart = rowStart + (maxChecksForBit+1);
  }

  // initialization
  // Build a matrix in which every row represents a check
  // and the elements, are the estimates for the bits contributing to this check.

  rowStart = 0;
  for (unsigned int bit=0; bit<numBits; bit++) {
    for (unsigned int index=1; index<=mapCols2Rows[rowStart]; index++) {
      oneDindex  = mapCols2Rows[rowStart +index];
      lambdaByCheckIndex[oneDindex] = lambda[bit];
    }
    rowStart = rowStart + (maxChecksForBit+1);
  }

  HANDLE_ERROR(hipMemcpy(dev_rSig, rSig, numBits * sizeof(float), hipMemcpyHostToDevice));

#ifdef INTERNAL_TIMINGS_4_DECODER
  float elapsedTime, partTimes;
  float  allTime = 0.0, nodeProcessingTime = 0.0, bitEstimateTime = 0.0, transposeTime = 0.0;
  hipEvent_t globalStart;
  HANDLE_ERROR(hipEventCreate(&globalStart));
  hipEvent_t startAt;
  HANDLE_ERROR(hipEventCreate(&startAt));
  hipEvent_t stopAt;
  HANDLE_ERROR(hipEventCreate(&stopAt));
  HANDLE_ERROR(hipEventRecord(globalStart, NULL));
#endif

  ////////////////////////////////////////////////////////////////////////////
  // Main iteration loop
  ////////////////////////////////////////////////////////////////////////////

  for(iterCounter=1;iterCounter<=maxIterations;iterCounter++) {

    HANDLE_ERROR(hipMemcpy(dev_eta, eta, nChecksByBits * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_lambdaByCheckIndex, lambdaByCheckIndex, nChecksByBits * sizeof(float), hipMemcpyHostToDevice));

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR(hipEventRecord(startAt, NULL));
#endif
    // checkNode Processing  (1536)
    checkNodeProcessing<<< (1535+NTHREADS)/NTHREADS,NTHREADS>>>(numChecks, maxBitsForCheck, dev_lambdaByCheckIndex, dev_eta);
    hipDeviceSynchronize();

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR( hipEventRecord(stopAt, NULL));
    HANDLE_ERROR( hipEventSynchronize(stopAt));
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, startAt, stopAt));
    nodeProcessingTime = nodeProcessingTime + elapsedTime;
#endif
    HANDLE_ERROR(hipMemcpy(eta, dev_eta, nChecksByBits * sizeof(float), hipMemcpyDeviceToHost));

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR(hipEventRecord(startAt, NULL));
#endif
    // Transpose  eta with rows == parity checks, to rows == bits
    rowStart = 0;
    for (unsigned int check=0; check<numChecks; check++) {
      for (unsigned int index=1; index<= mapRows2Cols[rowStart]; index++) {
        oneDindex = mapRows2Cols[rowStart + index];
        etaByBitIndex[oneDindex] = eta[rowStart + index];
      }
      rowStart = rowStart + (maxBitsForCheck+1);
    }

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR( hipEventRecord(stopAt, NULL));
    HANDLE_ERROR( hipEventSynchronize(stopAt));
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, startAt, stopAt));
    transposeTime = transposeTime + elapsedTime;
#endif

    // bit estimates update
    HANDLE_ERROR(hipMemcpy(dev_etaByBitIndex, etaByBitIndex, nBitsByChecks * sizeof(float), hipMemcpyHostToDevice));
#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR(hipEventRecord(startAt, NULL));
#endif
    bitEstimates<<<(2560+NTHREADS)/NTHREADS,NTHREADS>>>(dev_rSig, dev_etaByBitIndex, dev_lambda, numBits,maxChecksForBit);
    hipDeviceSynchronize();
#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR( hipEventRecord(stopAt, NULL));
    HANDLE_ERROR( hipEventSynchronize(stopAt));
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, startAt, stopAt));
    bitEstimateTime = bitEstimateTime + elapsedTime;
#endif
    HANDLE_ERROR(hipMemcpy(lambda, dev_lambda, numBits * sizeof(float), hipMemcpyDeviceToHost));

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR(hipEventRecord(startAt, NULL));
#endif
    // Transpose  lambda with rows == bits, to rows == parity checks
    rowStart = 0;
    for (unsigned int n=0; n<numBits; n++) {
      decision[n] = (lambda[n] >= 0) ? 1 : 0;
      estimates[n] = lambda[n];
      for (unsigned int index=1; index<=mapCols2Rows[rowStart]; index++) {
        oneDindex  = mapCols2Rows[rowStart + index];
        lambdaByCheckIndex[oneDindex] = lambda[n];
        cHat[oneDindex] = decision[n];
      }
      rowStart = rowStart + (maxChecksForBit+1);
    }
#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR( hipEventRecord(stopAt, NULL));
    HANDLE_ERROR( hipEventSynchronize(stopAt));
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, startAt, stopAt));
    transposeTime = transposeTime + elapsedTime;
#endif

    // Check for correct decoding.
    rowStart = 0;
    allChecksPassed = true;
    for (unsigned int check=0; check<numChecks; check++) {
      unsigned int sum = 0;
      for (unsigned int index=1; index<= cHat[rowStart]; index++) {sum = sum + cHat[rowStart + index];}
      if ((sum % 2) != 0 ) {
        allChecksPassed = false;
        break;
      }
      rowStart = rowStart + maxBitsForCheck+1;
    }
    if (allChecksPassed) {
      break;}
  }

  if(allChecksPassed == false) {
    // printf("Decoding failure after %d iterations\n", iterCounter);
  } else {
    // Print a status message on the iteration loop
    // printf("Success at %i iterations\n",iterCounter);
  }

#ifdef INTERNAL_TIMINGS_4_DECODER
  HANDLE_ERROR( hipEventRecord(stopAt, NULL));
  HANDLE_ERROR( hipEventSynchronize(stopAt));
  HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, globalStart, stopAt));
  allTime = elapsedTime;
  partTimes = nodeProcessingTime + bitEstimateTime + transposeTime;

  printf("\n");
  printf ("Total Time      : %.1f microsec\n", 1000*allTime);
  printf ("node processing : %.1f microsec (%.2f%)\n", 1000*nodeProcessingTime, 100 *nodeProcessingTime/allTime);
  printf ("bit estimates   : %.1f microsec (%.2f%)\n", 1000*bitEstimateTime, 100 * bitEstimateTime/allTime);
  printf ("transpose       : %.1f microsec (%.2f%)\n", 1000*transposeTime, 100 * transposeTime/allTime);
  printf ("Other???        : %.1f microsec (%.2f%)\n", 1000*(allTime - partTimes) , 100 * (allTime - partTimes)/allTime);
  printf("\n");
#endif

  return (iterCounter);
}
