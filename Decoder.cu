// Based on the Eric's C-code implementation of ldpc decode.
//
#include <cstdio>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <time.h>

#include "GPUincludes.h"
#include "LDPC.h"

#define NTHREADS   64
#define CNP_THREADS   20  // checkNodeProcessing threads

unsigned int nChecksByBits;
unsigned int nBitsByChecks;

bundleElt *eta;
bundleElt *etaByBitIndex;
bundleElt *lambdaByCheckIndex;
bundleElt *cHat;
bundleElt *parityBits;
bundleElt paritySum;

bundleElt *dev_rSig;
bundleElt *dev_eta;
bundleElt *dev_lambda;
bundleElt *dev_etaByBitIndex;
bundleElt *dev_lambdaByCheckIndex;
bundleElt *dev_cHat;
bundleElt *dev_parityBits;

unsigned int *dev_mapRC;
unsigned int *dev_mapCR;

size_t temp_storage_bytes;
int* temp_storage=NULL;

void initLdpcDecoder  (H_matrix *hmat, unsigned int nBundles) {

  unsigned int *mapRows2Cols = hmat->mapRows2Cols;
  unsigned int *mapCols2Rows = hmat->mapCols2Rows;
  unsigned int numBits = hmat->numBits;
  unsigned int numChecks = hmat->numChecks;
  unsigned int maxBitsPerCheck = hmat->maxBitsPerCheck;
  unsigned int maxChecksPerBit = hmat->maxChecksPerBit;

  bundleElt numContributorsBE;
  unsigned int bundleAddr;
  unsigned int nChecksByBits = numChecks*(maxBitsPerCheck+1);
  unsigned int nBitsByChecks = numBits*(maxChecksPerBit+1);

  HANDLE_ERROR( hipHostMalloc((void**) &eta, nChecksByBits* nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipHostMalloc((void**) & etaByBitIndex, nBitsByChecks * nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipHostMalloc((void**) & lambdaByCheckIndex, nChecksByBits* nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipHostMalloc((void**) & cHat, nChecksByBits* nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipHostMalloc((void**) & parityBits, numChecks * nBundles*sizeof(bundleElt)));

  HANDLE_ERROR( hipMalloc( (void**)&dev_rSig, numBits * nBundles*sizeof(bundleElt) ));
  HANDLE_ERROR( hipMalloc( (void**)&dev_eta, nChecksByBits * nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_lambda, numBits * nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_etaByBitIndex,  nBitsByChecks * nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_lambdaByCheckIndex, nChecksByBits * nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_mapRC, nChecksByBits * sizeof(unsigned int)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_mapCR, nBitsByChecks * sizeof(unsigned int)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_cHat, nChecksByBits * nBundles*sizeof(bundleElt)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_parityBits, numChecks * nBundles*sizeof(bundleElt)));

  HANDLE_ERROR(hipMemcpy(dev_mapRC, mapRows2Cols, nChecksByBits * sizeof(unsigned int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_mapCR, mapCols2Rows, nBitsByChecks * sizeof(unsigned int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cHat, cHat, nChecksByBits * nBundles*sizeof(bundleElt), hipMemcpyHostToDevice));

  memset(eta, 0, nChecksByBits*nBundles*sizeof(eta[0]));
  memset(etaByBitIndex, 0, nBitsByChecks*nBundles*sizeof(etaByBitIndex[0]));
  memset(lambdaByCheckIndex, 0, nChecksByBits*nBundles*sizeof(lambdaByCheckIndex[0]));

  // All matrices are stored in column order.
  // For eta and lambdaByCheckIndex, each column represents a Check node.
  // There are maxBitsPerCheck+1 rows.
  // row 0 always contains the number of contributors for this check node.
  for (unsigned int check=0; check<numChecks; check++) {
      numContributorsBE = make_bundleElt((float)mapRows2Cols[check]);
      for (unsigned int bundle=0; bundle  < nBundles; bundle++) {
        bundleAddr = bundle * nChecksByBits + check;
        eta[bundleAddr] = numContributorsBE;
        lambdaByCheckIndex[bundleAddr] = numContributorsBE;
        cHat[bundleAddr] = numContributorsBE;
      }
  }
  // Need to have row 0 (see preceding code segment) in lambdaByCheckIndex and cHat into device memory, now.
  // For each new record, these device memory matrices are updated with a kernel
  HANDLE_ERROR(hipMemcpy(dev_lambdaByCheckIndex, lambdaByCheckIndex, nChecksByBits * nBundles*sizeof(bundleElt), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cHat, cHat, nChecksByBits * nBundles*sizeof(bundleElt), hipMemcpyHostToDevice));

  // For etaByBitIndex, each column corresponds to a bit node.
  // row 0, contains the number of contributors for this bit.
  for (unsigned int bit=0; bit<numBits; bit++) {
    etaByBitIndex[bit] = make_bundleElt((float)mapCols2Rows[bit]);
  }
}

int ldpcDecoderWithInit (H_matrix *hmat, bundleElt *rSig, unsigned int  maxIterations, unsigned int *decision, bundleElt *estimates, unsigned int nBundles) {

  unsigned int numBits = hmat->numBits;
  unsigned int numChecks = hmat->numChecks;
  unsigned int maxBitsPerCheck = hmat->maxBitsPerCheck;
  unsigned int maxChecksPerBit = hmat->maxChecksPerBit;
  unsigned int nChecksByBits = numChecks*(maxBitsPerCheck+1);
  unsigned int nBitsByChecks = numBits*(maxChecksPerBit+1);

  unsigned int iterCounter;
  bool allChecksPassed = false;
  unsigned int bundleBase;
  unsigned int successCount;
  unsigned int returnVal;

  HANDLE_ERROR(hipMemcpy(dev_rSig, rSig, numBits * nBundles*sizeof(bundleElt), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_etaByBitIndex, etaByBitIndex, nBitsByChecks * nBundles*sizeof(bundleElt), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_eta, eta, nChecksByBits * nBundles*sizeof(bundleElt), hipMemcpyHostToDevice));
  copyBitsToCheckmatrix<<<numBits*nBundles, NTHREADS>>>(dev_mapCR, dev_rSig, dev_lambdaByCheckIndex, numBits, maxChecksPerBit,
                                                        nChecksByBits, nBitsByChecks, nBundles);

  ////////////////////////////////////////////////////////////////////////////
  // Main iteration loop
  ////////////////////////////////////////////////////////////////////////////

  for(iterCounter=1;iterCounter<=maxIterations;iterCounter++) {
    checkNodeProcessingOptimalBlock <<<numChecks*nBundles, CNP_THREADS>>>
      (numChecks, maxBitsPerCheck, dev_lambdaByCheckIndex, dev_eta, dev_mapRC, dev_etaByBitIndex,
       nChecksByBits, nBitsByChecks, nBundles);

    bitEstimates<<<(numBits*nBundles)/NTHREADS+1,NTHREADS>>>
      (dev_rSig, dev_etaByBitIndex, dev_lambdaByCheckIndex, dev_cHat, dev_mapCR, numBits,maxChecksPerBit,
       nChecksByBits, nBitsByChecks, nBundles);

    calcParityBits <<<(numChecks*nBundles)/NTHREADS+1, NTHREADS>>>
      (dev_cHat, dev_parityBits, numChecks, maxBitsPerCheck,
       nChecksByBits, nBundles);

    allChecksPassed = true;

    //  The cpu is slightly faster than GPU DeviceReduce  to determine if any paritycheck is non-zero.
    HANDLE_ERROR(hipMemcpy(parityBits, dev_parityBits, numChecks*nBundles*sizeof(bundleElt),hipMemcpyDeviceToHost));
    paritySum = make_bundleElt(0.0);
    // We loop over parity checks for all bundles (in a single loop here)
    for (unsigned int check=0; check < numChecks*nBundles; check++) {
      for (unsigned int slot=0; slot< SLOTS_PER_ELT; slot++) if ((int)parityBits[check].s[slot] != 0) allChecksPassed = false;
      if (! allChecksPassed) break;
    }
    if (allChecksPassed) break;
  }
  // Return our best guess.
  // if iterCounter < maxIterations, then successful.
  successCount = 0;
  for (unsigned int bundle=0; bundle  < nBundles; bundle++) {
    bundleBase = bundle* numChecks;
    paritySum = make_bundleElt(0.0);
    for (unsigned int check=0; check < numChecks; check++) paritySum += parityBits[bundleBase + check];
    for (unsigned int slot=0; slot< SLOTS_PER_ELT; slot++) if ((int)paritySum.s[slot] == 0) successCount++;
  }

  returnVal = (successCount << 8) + iterCounter;
  return (returnVal);
}
