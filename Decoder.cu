#include "hip/hip_runtime.h"
// #define INTERNAL_TIMINGS_4_DECODER ON

// Based on the Eric's C-code implementation of ldpc decode.
//
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <time.h>

#include "GPUincludes.h"

// #define MIN(a,b)  (((a) < (b)) ? (a) : (b))
#define ABS(a)  (((a) < (0)) ? (-(a)) : (a))
#define MAX_ETA                1e6
#define MIN_TANH_MAGNITUDE     1e-10
#define SCALE_FACTOR           0.75

#define NTHREADS   16
#define CNP_THREADS   20  // checkNodeProcessing threads

__global__ void
checkNodeProcessingOptimal (unsigned int numChecks, unsigned int maxBitsForCheck,
                            float *lambdaByCheckIndex, float *eta);
__global__ void
checkNodeProcessingOptimalBlock (unsigned int numChecks, unsigned int maxBitsForCheck,
                                 float *lambdaByCheckIndex, float *eta);

__global__ void
checkNodeProcessingMinSum (unsigned int numChecks, unsigned int maxBitsForCheck,
                           float *lambdaByCheckIndex, float *eta);

__global__ void
checkNodeProcessingMinSumBlock (unsigned int numChecks, unsigned int maxBitsForCheck,
                                float *lambdaByCheckIndex, float *eta);


__global__ void
checkNodeProcessingOptimalNaive (unsigned int numChecks, unsigned int maxBitsForCheck,
                            float *lambdaByCheckIndex, float *eta) {
  unsigned int m;
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int thisRowLength, thisRowStart, currentIndex;
  float value, product;
  float rowVals[128];

  if (tid < numChecks) {
    m = tid;
    thisRowStart = m * (maxBitsForCheck+1);
    thisRowLength = eta[thisRowStart];

    // Optimal solution using tanh
    // Each thread processes an entire row.

    // compute the tanh values, and temporarily store back into eta
    for (unsigned int n=1; n<= thisRowLength ; n++) {
      currentIndex = thisRowStart+n;
      eta[currentIndex] = tanhf ((eta[currentIndex] - lambdaByCheckIndex[currentIndex]) / 2.0);
    }

    // Compute the product of the other tanh terms for each non-zero elements.
    for (unsigned int n=1; n<= thisRowLength ; n++) {
      product = 1.0;
      for (unsigned int newvar=1; newvar<= thisRowLength; newvar++) {
        if (newvar != n) product=product* eta[thisRowStart+newvar];
      }
      value = -2 *atanhf(product);
      value = (value > MAX_ETA)? MAX_ETA : value;
      value = (value < -MAX_ETA)? -MAX_ETA : value;
      rowVals[n] =  value;
    }
    for (unsigned int n=1; n<= thisRowLength ; n++) { eta[thisRowStart+n] = rowVals[n];}
  }
}

__global__ void
checkNodeProcessingOptimalCheckZero (unsigned int numChecks, unsigned int maxBitsForCheck,
                            float *lambdaByCheckIndex, float *eta) {
  unsigned int m, indexOfZero;
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int thisRowLength, thisRowStart, currentIndex;
  float value, product, arg;
  float rowVals[128];
  int nzeros;

  if (tid < numChecks) {
    nzeros=0;
    indexOfZero = 0;
    m = tid;
    thisRowStart = m * (maxBitsForCheck+1);
    thisRowLength = eta[thisRowStart];
    product = 1.0;

    // Optimal solution using tanh
    // Each thread processes an entire row.
    // Compute the product of the tanh terms for all non-zero elements.
    for (unsigned int n=1; n<= thisRowLength ; n++) {
      currentIndex = thisRowStart+n;
      value =  tanhf ((eta[currentIndex] - lambdaByCheckIndex[currentIndex]) / 2.0);
      rowVals[n] = value;
      if (value == 0.0) {
        nzeros++;
        indexOfZero = n;
      } else {
        product =  product * value;
      }
    }

    // Now, set the value for each element in this row.
    // If there are 2 or more zero's, then the product without
    // any single value will still be zero.
    if (nzeros > 1) {
      for (unsigned int n=1; n<= thisRowLength; n++) {
        currentIndex = thisRowStart+n;
        eta[currentIndex] =  0.0;
      }
    } else if (nzeros == 1) {
      for (unsigned int n=1; n<= thisRowLength; n++) {
        currentIndex = thisRowStart+n;
        eta[currentIndex] =  (n == indexOfZero)?product : 0.0;
      }
    } else {
      for (unsigned int n=1; n<= thisRowLength; n++) {
        currentIndex = thisRowStart+n;
        arg = product/rowVals[n];
        value = -2 *atanhf(arg);
        value = (value > MAX_ETA)? MAX_ETA : value;
        value = (value < -MAX_ETA)? -MAX_ETA : value;
        eta[currentIndex] =  value;
      }
    }
  }
}


__global__ void
bitEstimates(float *rSig, float *etaByBitIndex, float *lambda,
             unsigned int numBits, unsigned int maxChecksForBit) {

  unsigned int n;
  unsigned int thisRowLength, thisRowStart;
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < numBits) {
    n = tid;
    float sum = rSig[n];
    thisRowStart = n*(maxChecksForBit+1);
    thisRowLength = etaByBitIndex[thisRowStart];
    for (unsigned int m=1; m<=thisRowLength; m++) {
      sum = sum + etaByBitIndex[thisRowStart +m];
    }
    lambda[n] = sum;
  }
}

// Transpose  checkRows matrix with rows == parity checks, to
//            bitRows matrix  with rows == bits
__global__ void
transposeRC (unsigned int* map, float *checkRows, float *bitRows,
             unsigned int numChecks, unsigned int maxBitsForCheck) {
  // index
  unsigned int check = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int rowStart;
  unsigned int cellIndex, oneDindex;

  if (check < numChecks) {
    rowStart = check * (maxBitsForCheck+1);
    for (unsigned int index=1; index<= map[rowStart]; index++) {
      cellIndex = rowStart + index;
      oneDindex = map[cellIndex];
      bitRows[oneDindex] = checkRows[cellIndex];
    }
  }
}

// copyBitsToCheckmatrix accepts a vector of the current bitEstimates
// and copies them into a checkRow matrix, where each row represents a check.
// It also generates a HardDecision copy of that output matrix checkRows.
__global__ void
copyBitsToCheckmatrix (unsigned int* map, float *bitEstimates, float *checkRows,
                       unsigned int *hd,
                       unsigned int numBits, unsigned int maxChecksForBit) {
  // index
  unsigned int bitIndex = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int rowStart = 0;
  unsigned int cellIndex, oneDindex;
  float thisBitEstimate;

  if (bitIndex < numBits) {
    rowStart = bitIndex * (maxChecksForBit+1);
    thisBitEstimate = bitEstimates[bitIndex];
    for (unsigned int index=1; index<= map[rowStart]; index++) {
      cellIndex = rowStart + index;
      oneDindex = map[cellIndex];
      checkRows[oneDindex] = thisBitEstimate;
      hd[oneDindex] = (thisBitEstimate >= 0) ? 1 : 0;
    }
  }
}

int ldpcDecoder (float *rSig, unsigned int numChecks, unsigned int numBits,
                 unsigned int maxBitsForCheck, unsigned int maxChecksForBit,
                 unsigned int *mapRows2Cols,
                 unsigned int *mapCols2Rows,
                 unsigned int maxIterations,
                 unsigned int *decision,
                 float *estimates) {

  // Number of elements in a checkRows matrix (matrix with rowNum = CheckIndex)
  unsigned int nChecksByBits = numChecks*(maxBitsForCheck+1);
  // Number of elements in a bitRows matrix (matrix with rowNum = BitIndex)
  unsigned int nBitsByChecks = numBits*(maxChecksForBit+1);

  float eta[nChecksByBits];
  float lambda[numBits];
  float etaByBitIndex[nBitsByChecks];
  float lambdaByCheckIndex[nChecksByBits];
  unsigned int cHat [nChecksByBits];

  unsigned int iterCounter;
  bool allChecksPassed = false;

  unsigned int oneDindex;
  unsigned int rowStart;
  unsigned int rowLength;

  float *dev_rSig;
  float *dev_eta;
  float *dev_lambda;
  float *dev_etaByBitIndex;
  float *dev_lambdaByCheckIndex;
  unsigned int *dev_cHat;

  unsigned int *dev_mapRC;
  unsigned int *dev_mapCR;

  HANDLE_ERROR( hipMalloc( (void**)&dev_rSig, numBits * sizeof(float) ));
  HANDLE_ERROR( hipMalloc( (void**)&dev_eta, nChecksByBits * sizeof(float)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_lambda, numBits * sizeof(float)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_etaByBitIndex,  nBitsByChecks * sizeof(float)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_lambdaByCheckIndex, nChecksByBits * sizeof(float)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_mapRC, nChecksByBits * sizeof(unsigned int)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_mapCR, nBitsByChecks * sizeof(unsigned int)));
  HANDLE_ERROR( hipMalloc( (void**)&dev_cHat, nChecksByBits * sizeof(unsigned int)));

  memcpy(lambda, rSig, numBits*sizeof(lambda[0]));
  memset(eta, 0, nChecksByBits*sizeof(eta[0]));
  memset(etaByBitIndex, 0, nBitsByChecks*sizeof(etaByBitIndex[0]));
  memset(lambdaByCheckIndex, 0, nChecksByBits*sizeof(lambdaByCheckIndex[0]));

  // Need to insert rowLengths into eta (and lambdaByCheckIndex)
  // with rows corresponding to parity checks.
  rowStart = 0;
  for (unsigned int check=0; check<numChecks; check++) {
    rowLength = mapRows2Cols[rowStart];
    eta[rowStart] = (float)rowLength;
    lambdaByCheckIndex[rowStart] = (float)rowLength;
    cHat[rowStart] = rowLength;
    rowStart = rowStart + (maxBitsForCheck+1);
  }

  // Need to insert rowLengths into etaByBitIndex
  rowStart = 0;
  for (unsigned int bit=0; bit<numBits; bit++) {
    etaByBitIndex[rowStart] = (float)mapCols2Rows[rowStart];
    rowStart = rowStart + (maxChecksForBit+1);
  }

  // initialization
  // Build a matrix in which every row represents a check
  // and the elements, are the estimates for the bits contributing to this check.

  rowStart = 0;
  for (unsigned int bit=0; bit<numBits; bit++) {
    for (unsigned int index=1; index<=mapCols2Rows[rowStart]; index++) {
      oneDindex  = mapCols2Rows[rowStart +index];
      lambdaByCheckIndex[oneDindex] = lambda[bit];
    }
    rowStart = rowStart + (maxChecksForBit+1);
  }

  HANDLE_ERROR(hipMemcpy(dev_rSig, rSig, numBits * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_mapRC, mapRows2Cols, nChecksByBits * sizeof(unsigned int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_mapCR, mapCols2Rows, nBitsByChecks * sizeof(unsigned int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_etaByBitIndex, etaByBitIndex, nBitsByChecks * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cHat, cHat, nChecksByBits * sizeof(unsigned int), hipMemcpyHostToDevice));

#ifdef INTERNAL_TIMINGS_4_DECODER
  float elapsedTime, partTimes;
  float  allTime = 0.0, nodeProcessingTime = 0.0, bitEstimateTime = 0.0, transposeTime = 0.0;
  hipEvent_t globalStart;
  HANDLE_ERROR(hipEventCreate(&globalStart));
  hipEvent_t startAt;
  HANDLE_ERROR(hipEventCreate(&startAt));
  hipEvent_t stopAt;
  HANDLE_ERROR(hipEventCreate(&stopAt));
  HANDLE_ERROR(hipEventRecord(globalStart, NULL));
#endif

  ////////////////////////////////////////////////////////////////////////////
  // Main iteration loop
  ////////////////////////////////////////////////////////////////////////////

  HANDLE_ERROR(hipMemcpy(dev_eta, eta, nChecksByBits * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_lambdaByCheckIndex, lambdaByCheckIndex, nChecksByBits * sizeof(float), hipMemcpyHostToDevice));

  for(iterCounter=1;iterCounter<=maxIterations;iterCounter++) {

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR(hipEventRecord(startAt, NULL));
#endif
    // checkNodeProcessingMinSum <<< (numChecks)/NTHREADS+1,NTHREADS>>>(numChecks, maxBitsForCheck, dev_lambdaByCheckIndex, dev_eta);
    // checkNodeProcessingMinSumBlock <<< numChecks,32>>>(numChecks, maxBitsForCheck, dev_lambdaByCheckIndex, dev_eta);
    //  1 thread per block is significantly slower than 2.  (23300 :: 26000 msec) though I think I'm using just 1.
    //  >2 does not help much more.
    // checkNodeProcessingOptimal <<<numChecks/2,2>>>(numChecks, maxBitsForCheck, dev_lambdaByCheckIndex, dev_eta);

    checkNodeProcessingOptimalBlock <<<numChecks, 32>>>(numChecks, maxBitsForCheck, dev_lambdaByCheckIndex, dev_eta);

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR( hipEventRecord(stopAt, NULL));
    HANDLE_ERROR( hipEventSynchronize(stopAt));
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, startAt, stopAt));
    nodeProcessingTime = nodeProcessingTime + elapsedTime;
#endif

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR(hipEventRecord(startAt, NULL));
#endif
    transposeRC<<<(numChecks)/NTHREADS+1,NTHREADS>>>(dev_mapRC, dev_eta, dev_etaByBitIndex, numChecks, maxBitsForCheck);

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR( hipEventRecord(stopAt, NULL));
    HANDLE_ERROR( hipEventSynchronize(stopAt));
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, startAt, stopAt));
    transposeTime = transposeTime + elapsedTime;
#endif

    // bit estimates update
#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR(hipEventRecord(startAt, NULL));
#endif
    bitEstimates<<<(numBits)/NTHREADS+1,NTHREADS>>>(dev_rSig, dev_etaByBitIndex, dev_lambda, numBits,maxChecksForBit);

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR( hipEventRecord(stopAt, NULL));
    HANDLE_ERROR( hipEventSynchronize(stopAt));
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, startAt, stopAt));
    bitEstimateTime = bitEstimateTime + elapsedTime;
#endif

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR(hipEventRecord(startAt, NULL));
#endif
    // This resembles the earlier transpose operation, and so
    // this time is accumulated with it.

    // copy lambda (current bit estimates) into
    // a checkMatrix (where each row represents a check
    copyBitsToCheckmatrix<<<(numBits)/NTHREADS+1,NTHREADS>>>(dev_mapCR, dev_lambda, dev_lambdaByCheckIndex,
                                                                 dev_cHat, numBits, maxChecksForBit);

    HANDLE_ERROR(hipMemcpy(cHat, dev_cHat, nChecksByBits * sizeof(unsigned int),hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cHat, dev_cHat, nChecksByBits * sizeof(unsigned int),hipMemcpyDeviceToHost));

#ifdef INTERNAL_TIMINGS_4_DECODER
    HANDLE_ERROR( hipEventRecord(stopAt, NULL));
    HANDLE_ERROR( hipEventSynchronize(stopAt));
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, startAt, stopAt));
    transposeTime = transposeTime + elapsedTime;
#endif

    //DEBUG
    // if( iterCounter == 1) {
    //   printf("rSig: %.2f,  %.2f,  %.2f,  %.2f,  %.2f,  %.2f\n",
    //          rSig[0],rSig[1],rSig[2],rSig[3],rSig[4],rSig[5]);
    // }

    // if( iterCounter < 10) {
    //   HANDLE_ERROR(hipMemcpy(lambda, dev_lambda, numBits * sizeof(float), hipMemcpyDeviceToHost));
    //   printf("Lambda  Iter: %i: %.2f,  %.2f,  %.2f,  %.2f,  %.2f,  %.2f\n",
    //           iterCounter, lambda[0],lambda[1],lambda[2],lambda[3],lambda[4],lambda[5]);
    // }

    // Check for correct decoding.
    rowStart = 0;
    allChecksPassed = true;
    for (unsigned int check=0; check<numChecks; check++) {
      unsigned int sum = 0;
      for (unsigned int index=1; index<= cHat[rowStart]; index++) {sum = sum + cHat[rowStart + index];}
      if ((sum % 2) != 0) {
        allChecksPassed = false;
        break;
      }
      rowStart = rowStart + maxBitsForCheck+1;
    }
    if (allChecksPassed) {break;}
  }

  if(allChecksPassed == false) {
    // printf("Decoding failure after %d iterations\n", iterCounter);
  } else {
    // Print a status message on the iteration loop
    // printf("Success at %i iterations\n",iterCounter);
  }


#ifdef INTERNAL_TIMINGS_4_DECODER
  HANDLE_ERROR( hipEventRecord(stopAt, NULL));
  HANDLE_ERROR( hipEventSynchronize(stopAt));
  HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, globalStart, stopAt));
  allTime = elapsedTime;
  partTimes = nodeProcessingTime + bitEstimateTime + transposeTime;

  printf("\n");
  printf ("Total Time      : %.1f microsec\n", 1000*allTime);
  printf ("node processing : %.1f microsec (%.2f%)\n", 1000*nodeProcessingTime, 100 *nodeProcessingTime/allTime);
  printf ("bit estimates   : %.1f microsec (%.2f%)\n", 1000*bitEstimateTime, 100 * bitEstimateTime/allTime);
  printf ("transpose       : %.1f microsec (%.2f%)\n", 1000*transposeTime, 100 * transposeTime/allTime);
  printf ("Other???        : %.1f microsec (%.2f%)\n", 1000*(allTime - partTimes) , 100 * (allTime - partTimes)/allTime);
  printf("\n");
#endif

  return (iterCounter);
}
