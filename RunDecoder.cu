#include "hip/hip_runtime.h"
#include <random>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <errno.h>
#include <thread>
#include <chrono>

#include <hip/hip_runtime_api.h>

#include "GPUincludes.h"
#include "LDPC.h"

//#include "loader_pool.h"
#include "fast_loader.h"
#include "decoder_pool.h"

#define HISTORY_LENGTH  20

int main (int argc, char **argv) {

  using clock = std::chrono::steady_clock;

  clock::time_point startTime;
  clock::time_point endTime;
  clock::duration allTime;

  int status;
  unsigned int numChecks, numBits;
  unsigned int numRowsW, numColsW, numParityBits, shiftRegLength;
  unsigned int numThreads, sigBufferLength;
  unsigned int *W_ROW_ROM;

  H_matrix *hmat = (H_matrix*) malloc(sizeof(H_matrix));

  int maxIterations;
  char H_Alist_File[256];
  char wROM_File[256];
  FILE *src;
  int errnum;
  unsigned int infoLeng, rnum, rdenom;
  float ebno;
  unsigned int how_many;
  float rNominal;
  float No, sigma2, lc;

  if (argc < 8) {
    printf("usage:  RunDecoder <infoLength> <r-numerator> <r-denominator> <ebno> <numpackets> <maxIterations> <# Threads>\n" );
    exit(-1);
  }
  infoLeng = atoi(argv[1]);
  rnum = atoi(argv[2]);
  rdenom = atoi(argv[3]);
  rNominal = float(rnum)/float(rdenom);
  ebno = atof(argv[4]);
  how_many = atoi(argv[5]);
  maxIterations = atoi(argv[6]);
  numThreads = atoi(argv[7]);

  sprintf(H_Alist_File, "./G_and_H_Matrices/H_%d%d_%d.alist", rnum, rdenom, infoLeng);
  sprintf(wROM_File, "./G_and_H_Matrices/W_ROW_ROM_%d%d_%d.binary", rnum, rdenom, infoLeng);


  // Noise variance and log-likelihood ratio (LLR) scale factor. Because
  // Ec = R*Eb is unity (i.e. we transmit +1's and -1's), then No = 1/(R*EbNo).
  No = 1/(rNominal * pow(10,(ebno/10)));
  sigma2 = No/2;
  // When r is scaled by Lc it results in precisely scaled LLRs
  lc = 4/No;

  status = ReadAlistFile(hmat, H_Alist_File);
  if ( status != 0) {
    printf ("Unable to read alist file: %s\n", H_Alist_File);
    exit(-1);
  }
  numBits = hmat->numBits;
  numChecks = hmat->numChecks;

  src = fopen(wROM_File, "r");
  if (src == NULL) {
    errnum = errno;
    printf("Value of errno: %d\n", errnum);
    perror("Error printed by perror");
    printf("Error opening file %s\n", wROM_File);
    return(EXIT_FAILURE);
  }

  fread(& numRowsW, sizeof(unsigned int), 1, src);
  fread(& numColsW, sizeof(unsigned int), 1, src);
  fread(& shiftRegLength, sizeof(unsigned int), 1, src);
  W_ROW_ROM = (unsigned int*) malloc(numRowsW * numColsW * sizeof( unsigned int));
  fread(W_ROW_ROM, sizeof(unsigned int), numRowsW * numColsW, src);
  numParityBits = numColsW;
  fclose(src);


  printf("parameters have been read.\n");
  printf("SLOTS_PER_ELT = %d\n", SLOTS_PER_ELT);
  printf("numBits = %i, numChecks = %i\n", numBits, numChecks);
  printf("infoLeng = %i, numParityBits = %i (%i), numBits = %i\n",
         infoLeng, numParityBits, infoLeng + numParityBits, numBits);
  printf("maxChecksPerBit = %i maxBitsPerCheck = %i\n", hmat->maxChecksPerBit, hmat->maxBitsPerCheck);
  printf("ebn0 = %f, sigma = %f\n", ebno, sigma2);

  // ///////////////////////////////////////////

  bundleElt *receivedSigs;
  bundleElt *decodedSigs;
  bundleElt zeroBE = make_bundleElt(0.0);
  unsigned int sigIndex;

  unsigned int successes = 0;
  unsigned int iterationSum = 0;
  int iters;

  /* noisy signal generation */
  unsigned int  seed = 163331;
  std::mt19937 generator(seed); //Standard mersenne_twister_engine
  std::uniform_real_distribution<> rDist(0, 1);

  std::normal_distribution<float> normDist(0.0, 1.0);
  unsigned int* infoWord;
  unsigned int* codeWord;
  float s, noise;
  float* receivedSig;
  bundleElt* bundle;
  bundleElt* preloads;

  infoWord = (unsigned int *)malloc(infoLeng * sizeof(unsigned int));
  codeWord = (unsigned int *)malloc((infoLeng + numParityBits) * sizeof(unsigned int));
  receivedSig = (float *)malloc(numBits * sizeof(float));

  int nBundles = 1000;
  int bundleStart;
  bundle = (bundleElt*) malloc(numBits * sizeof(bundleElt));
  preloads = (bundleElt*) malloc(nBundles * numBits * sizeof(bundleElt));

  for (int bundleIndex = 0; bundleIndex < nBundles; bundleIndex++) {
    bundleStart = bundleIndex * numBits;
    for (unsigned int slot=0; slot < SLOTS_PER_ELT; slot++) {
      for (unsigned int j=0; j < infoLeng; j++) infoWord[j] = (0.5 >= rDist(generator))? 1:0;
      ldpcEncoder(infoWord, W_ROW_ROM, infoLeng, numRowsW, numColsW, shiftRegLength, codeWord);

      for (unsigned int j=0; j < (infoLeng+numParityBits) ; j++) {
        s     = 2*float(codeWord[j]) - 1;
        noise = sqrt(sigma2) * normDist(generator);
        receivedSig[j]  = lc*(s + noise);
      }
      // The LDPC codes are punctured, so the r we feed to the decoder is
      // longer than the r we got from the channel. The punctured positions are filled in as zeros
      for (unsigned int j=(infoLeng+numParityBits); j<numBits; j++) receivedSig[j] = 0.0;
      for (unsigned int j=0; j < numBits; j++ ) bundle[j].s[slot] = receivedSig[j];
    }
    for (unsigned int j=0; j < numBits; j++) preloads[bundleStart+j] = bundle[j];
  }
  printf ("Encoding complete.\n");

  // An ugly way to intialize variable allTime (accumulated interesting time) to zero.
  startTime = clock::now();
  allTime = startTime - startTime;

  sigBufferLength = 2 * numThreads;
  receivedSigs = (bundleElt*) malloc(sigBufferLength * numBits * sizeof(bundleElt));
  decodedSigs  = (bundleElt*) malloc(sigBufferLength * numBits * sizeof(bundleElt));

  std::vector<Tpkt> buffer;
  buffer.reserve(sigBufferLength);

  DecoderPool* decoders = new DecoderPool(hmat, maxIterations, numThreads);
  //  LoaderPool* pktLoader = new LoaderPool(infoLeng, numBits, numParityBits, W_ROW_ROM, numRowsW, numColsW, shiftRegLength, sigma2, lc);
  FastLoader* pktLoader = new FastLoader(preloads, nBundles, numBits);

  for (unsigned int i=0; i< sigBufferLength; i++) {
    sigIndex = i* numBits;
    buffer.emplace_back(&receivedSigs[sigIndex],  & decodedSigs[sigIndex]);
    buffer[i].state = LOADING;
    pktLoader->schedule_job(&buffer[i]);
    std::this_thread::sleep_for(std::chrono::milliseconds(10));
  }

  unsigned int pktsDecoded = 0;
  startTime = clock::now();

  hipProfilerStart();

  while (pktsDecoded < how_many) {
    for (unsigned int i=0; i< sigBufferLength; i++) {
      switch(buffer[i].state) {
      case LOADING :
        if (buffer[i].loadStamp != 0 ) {
          buffer[i].loadStamp = 0;
          buffer[i].state = DECODING;
          decoders->schedule_job(&buffer[i]);
        }
        break;
      case DECODING :
        if (buffer[i].decodeStamp != 0 ) {
          iters = buffer[i].decodeStamp;
          successes += iters >> 8;
          iters = iters & 0xff ;
          iterationSum = iterationSum + iters;
          pktsDecoded += SLOTS_PER_ELT;
          if ((pktsDecoded % 10000) == 0) {
            printf (" .");
            fflush(stdout);
          }
          buffer[i].decodeStamp = 0;
          buffer[i].state = LOADING;
          pktLoader->schedule_job(&buffer[i]);
        }
        break;
      }
    }
  }
  hipProfilerStop();

  endTime = clock::now();
  allTime = endTime - startTime;

  printf("\n");
  delete pktLoader;
  delete decoders;

  printf("%i msec to decode %i packets.\n",std::chrono::duration_cast<std::chrono::milliseconds>(allTime).count(),pktsDecoded);
  printf(" %i Successes out of %i packets. (%.2f%%)\n", successes, pktsDecoded, 100.0 * successes/ pktsDecoded);
  printf("Information rate: %.2f Mbps\n", successes * infoLeng / (1000.0 * std::chrono::duration_cast<std::chrono::milliseconds>(allTime).count()));
  // SLOTS_PER_ELT packets are handled in each iteration, so...
  iterationSum = iterationSum * SLOTS_PER_ELT;
  printf(" %i cumulative iterations, or about %.1f per packet.\n", iterationSum, iterationSum/(float)pktsDecoded);
  //  printf("Number of iterations for the first few packets:  ");
  //  for (unsigned int i=1; i<= MIN(pktsDecoded, HISTORY_LENGTH); i++) {printf(" %i", itersHistory[i]);}
  //  printf ("\n");

  hipDeviceReset();
}
