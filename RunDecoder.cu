#include <math.h>
#include <string.h>

#include "mat.h"
#include "matrix.h"

#include "GPUincludes.h"

#define MAXITERATIONS  200

int ldpcDecoder (float *rSig, unsigned int numChecks, unsigned int numBits,
                 unsigned int *bitsForCheck, unsigned int *checksForBit,
                 int maxBitsForCheck, int maxChecksForBit,
                 unsigned int *mapRows2Cols, unsigned int *mapCols2Rows,
                 unsigned int maxIterations,
                 int *decision);

int main () {
  MATFile *pmat;
  mxArray *p1, *p2, *p3, *p4, *p5, *p6, *p7, *p8;
  unsigned int numChecks, numBits, maxBitsForCheck, maxChecksForBit, infoLength;
  unsigned int *bitsForCheck, *checksForBit;
  unsigned int  *mapRows2Cols, *mapCols2Rows;
  double *receivedSigs;
  unsigned int sigLength, numSigs;
  //  char filename[] ="~/Projects/CodedAPSK/matlabDecoder/Data/Signals/sig_2.5.mat";
  char filename[] ="~/APSK/Data/Signals/sig_2.5_short.mat";

  pmat = matOpen(filename, "r");
  if (pmat == NULL) {
    printf("Error opening file %s\n", filename);
    return(EXIT_FAILURE);
  }

  p1 = matGetVariable(pmat, "bitsForCheck");
  bitsForCheck = (unsigned int *)mxGetData(p1);
  numChecks = (unsigned int)mxGetM(p1);

  p2 = matGetVariable(pmat, "checksForBit");
  checksForBit = (unsigned int *)mxGetData(p2);
  numBits = (unsigned int)mxGetM(p2);

  p3 = matGetVariable(pmat, "maxBitsForCheck");
  maxBitsForCheck = (unsigned int)mxGetScalar(p3);

  p4 = matGetVariable(pmat, "maxChecksForBit");
  maxChecksForBit = (unsigned int)mxGetScalar(p4);

  p5 = matGetVariable(pmat, "infoLength");
  infoLength = (unsigned int)mxGetScalar(p5);

  p6 = matGetVariable(pmat, "mapRows2Cols");
  mapRows2Cols  = (unsigned int *)mxGetData(p6);

  p7 = matGetVariable(pmat, "mapCols2Rows");
  mapCols2Rows  = (unsigned int *)mxGetData(p7);

  p8 = matGetVariable(pmat, "receivedSigs");
  receivedSigs = mxGetPr(p8);
  sigLength = mxGetM(p8);
  numSigs = mxGetN(p8);

  matClose(pmat);
  // ///////////////////////////////////////////

  int decision[sigLength];
  int niters[sigLength];
  float rSig[sigLength];
  unsigned int sigStartIndex;

  int successes = 0;
  int iterationSum = 0;
  int numreps = 1;

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start;
  HANDLE_ERROR(hipEventCreate(&start));
  hipEvent_t stop;
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, NULL));

  for (int reps = 0; reps < numreps; reps++) {
    for (unsigned int i=0; i<numSigs; i++) {
      sigStartIndex = i * sigLength;
      for (unsigned int j=0; j<sigLength; j++) {rSig[j] =  (float)receivedSigs[sigStartIndex+j];   }
      niters[i] = ldpcDecoder(rSig, numChecks, numBits, bitsForCheck, checksForBit,
                              maxBitsForCheck, maxChecksForBit, mapRows2Cols, mapCols2Rows, MAXITERATIONS, decision);
      if (niters[i] < MAXITERATIONS) {successes++;}
      iterationSum = iterationSum + niters[i];  }
  }
  // Record the stop event
  HANDLE_ERROR( hipEventRecord(stop, NULL));
  // Wait for the stop event to complete
  HANDLE_ERROR( hipEventSynchronize(stop));
  float msecTotal = 0.0f;
  HANDLE_ERROR( hipEventElapsedTime(&msecTotal, start, stop));
  printf("%f msec to decode %i packets.\n", msecTotal, numSigs* numreps);

  printf(" %i Successes out of %i inputs.\n", successes, numSigs);
  printf(" %i cumulative iterations, or about %.1f per packet.\n", iterationSum, iterationSum/(float)numSigs);
  printf("Number of iterations for the first few packets:  ");
  for (int i=0; i<10; i++) {printf(" %i", niters[i]);}
  printf ("\n");
}
