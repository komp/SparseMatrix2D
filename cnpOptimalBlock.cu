#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <time.h>

#include "bundleElt.h"

#define MAX_ETA                1e6
#define MIN_TANH_MAGNITUDE     1e-10

// NOTE:  pragma unroll before the loops did not improve performance
// For implementation,
// I had to make the loop termination index constant; and wrap the
//  the loop body in a if (n < thisRowLength) ...
__global__ void
checkNodeProcessingOptimalBlock (unsigned int numChecks, unsigned int maxBitsForCheck,
                                 bundleElt *lambdaByCheckIndex, bundleElt *eta, unsigned int* mapRows2Cols,
                                 bundleElt *etaByBitIndex) {

  unsigned int m, n;
  unsigned int thisRowLength, currentIndex;
  bundleElt arg, value;

  m = blockIdx.x;
  n = threadIdx.x + 1;
  if (m < numChecks) {
    __shared__ bundleElt rowVals[128];

    thisRowLength = (int) ONEVAL(eta[m]);
    if (n <= thisRowLength) {
      currentIndex = m + (n* numChecks);
      arg =  (eta[currentIndex] - lambdaByCheckIndex[currentIndex]) / 2.0;
      value.x = tanhf(arg.x);
      value.y = tanhf(arg.y);
      value.z = tanhf(arg.z);
      value.w = tanhf(arg.w);
      if (value.x == 0.0) {value.x = MIN_TANH_MAGNITUDE;}
      if (value.y == 0.0) {value.y = MIN_TANH_MAGNITUDE;}
      if (value.z == 0.0) {value.z = MIN_TANH_MAGNITUDE;}
      if (value.w == 0.0) {value.w = MIN_TANH_MAGNITUDE;}
      rowVals[n] = value;
      __syncthreads();

      // Using JUST thread 0 to compute the product of all terms.
      // Storing it in the shared location  rowVals[0]
      if (threadIdx.x == 0) {
        rowVals[0] = makeBundleElt(1.0);
        for (unsigned int j=1; j<= thisRowLength; j++) rowVals[0] *= rowVals[j];
      }
      __syncthreads();

      // value = -2 *atanhf(rowVals[0]/rowVals[n]);
      arg = rowVals[0]/rowVals[n];
      value.x = -2 * atanhf(arg.x);
      value.y = -2 * atanhf(arg.y);
      value.z = -2 * atanhf(arg.z);
      value.w = -2 * atanhf(arg.w);

      value = clamp(value, -MAX_ETA, MAX_ETA);
      eta[currentIndex] =  value;
      etaByBitIndex[ mapRows2Cols[currentIndex] ] = value;
    }
  }
}
